#include "hip/hip_runtime.h"
//
// Created by jarry_goon on 2024-02-16.
//

#include "YOLO.cuh"
#include "cuda_kernel_funtion.cuh"

#include <opencv2/cudaimgproc.hpp>
#include <opencv2/cudawarping.hpp>
#include <opencv2/cudaarithm.hpp>

#include <hip/hip_fp16.h>

#include <iostream>

#define NMS_THRESHOLD 0.7f

#define SIZE_FLOAT  (int) 4 // = sizeof(float)
#define SIZE_HALF   (int) 2 // = sizeof(half)

YOLO::YOLO(const std::string& model_path)
    : model(model_path),
      mode(NONE),
      input_bind_idx(-1),
      output0_bind_idx(-1),
      output1_bind_idx(-1),
      aspect_ratio(0.f)
{
    std::vector<LayerInfo> layers;

    int num_layers;

    // AI 모델 입출력 레이어 정보 추출
    layers     = model.get_IO_layers();
    num_layers = layers.size();

    if(num_layers > 3)
    {
        fprintf(stderr, "ERROR: It is not YOLOv8 model. Check the model.\n");

        return;
    }

    // 입출력 레이어 분류
    for(int i = 0; i < num_layers; i++)
    {
        // YOLO에서 입력 레이어는 모델 관계없이 img 1개
        if(auto [layer_name, io_mode, data_type, dims] = layers[i];
            io_mode == nvinfer1::TensorIOMode::kINPUT)
        {
            input_layer_size.width  = static_cast<int>(dims.d[3]);
            input_layer_size.height = static_cast<int>(dims.d[2]);
            img_datatype            = data_type;
            input_bind_idx          = i;
        }
        else if(layer_name == "output0" && io_mode == nvinfer1::TensorIOMode::kOUTPUT)
        {
            output0_dims     = dims;
            output0_bind_idx = i;
        }
        else if(layer_name == "output1" && io_mode == nvinfer1::TensorIOMode::kOUTPUT)
        {
            output1_dims     = dims;
            output1_bind_idx = i;
        }
    }

    // 입력 레이어가 1개 초과일 경우 YOLO 모델이 아니기 때문에 종료
    if(input_bind_idx != 0)
    {
        fprintf(stderr, "ERROR: It is not YOLOv8 model. Check the model.\n");

        return;
    }

    // 입출력 레이어 특성 기준으로 모드 선정
    if(layers.size() < 3)
        mode = DETECTION;
    else
        mode = SEGMENTATION;
}

bool YOLO::predict(const cv::Mat& img, const float threshold, const uint32_t max_object)
{
    cv::cuda::GpuMat resized_img;
    cv::cuda::GpuMat blob_img;

    /*------------------------------------------------ 전처리 ---------------------------------------------------------*/

    // 1. 이미지 크기 변환
    // 모델 입력 기준과 크기가 맞지 않는 경우 크기와 비율이 모두 일치하는 이미지로 변환
    if(img_size.height != input_layer_size.height || img_size.width != input_layer_size.width)
        resize_img(img, resized_img);
        // 사이즈가 같을 경우 이미지 크기 유지
    else
        resized_img.upload(img);

    // 2. 이미지 BGR에서 RGB로 변경
    cv::cuda::cvtColor(resized_img, resized_img, cv::COLOR_BGR2RGB);

    // 3. HWC에서 CHW로 변환
    blob(resized_img, blob_img);

    // 4. 이미지 데이터를 부동소숫점 형태로 바꾸고 Normalize
    blob_img.convertTo(blob_img, CV_32FC3, 1.f / 255.f);

    // 5. 후처리에 사용할 변수 저장
    img_size     = img.size();    // 원본 이미지 크기 저장
    aspect_ratio = 1.f / std::min(static_cast<float>(input_layer_size.width) / static_cast<float>(img.cols),
                                  static_cast<float>(input_layer_size.height) / static_cast<float>(img.rows));

    // 6. 모델의 역할에 맞는 추정 프로세스 실행
    switch(mode)
    {
        case DETECTION:
            return detection(blob_img.data, threshold, max_object);

        case SEGMENTATION:
            return segmentation(blob_img.data, threshold, max_object);

        default:
        case NONE:
            fprintf(stderr, "ERROR: Can not predict.\n");

            return false;
    }
}

void YOLO::resize_img(const cv::Mat& img, cv::cuda::GpuMat& dst) const
{
    cv::cuda::GpuMat img_device;
    cv::cuda::GpuMat unpadding_resized_img;

    cv::Rect roi;

    int unpadding_width;
    int unpadding_height;

    float ratio;

    // 사이즈를 줄이고 비율을 맞췄을 때 남는 부분을 채우기 위해서 검은색을 기본으로 설정
    dst = cv::cuda::GpuMat(input_layer_size, img.type(), cv::Scalar::all(0));

    // 이미지를 GPU에 업로드
    img_device.upload(img);

    // 가로 세로 비율 계산, 가장 긴 값에 맞춤
    ratio = std::min(static_cast<float>(input_layer_size.width) / static_cast<float>(img.cols),
                     static_cast<float>(input_layer_size.height) / static_cast<float>(img.rows));

    // 스케일링된 이미지 가로 세로 길이
    unpadding_width  = static_cast<int>(ratio * static_cast<float>(img.cols));
    unpadding_height = static_cast<int>(ratio * static_cast<float>(img.rows));

    // 패딩 되지 않은 스케일링된 이미지 생성
    unpadding_resized_img = cv::cuda::GpuMat(unpadding_height, unpadding_width, CV_8UC3);
    cv::cuda::resize(img_device, unpadding_resized_img, unpadding_resized_img.size());

    // 스케일링된 이미지를 입력 레이어 특성에 맞게 패딩(Padding)
    roi = cv::Rect(0, 0, unpadding_width, unpadding_height);
    unpadding_resized_img.copyTo(dst(roi));

    img_device.release();
}

void YOLO::blob(const cv::cuda::GpuMat& img, cv::cuda::GpuMat& dst) const
{
    const int width = input_layer_size.height * input_layer_size.width;

    dst = cv::cuda::GpuMat(1, width, CV_8UC3);

    std::vector<cv::cuda::GpuMat> input_channels{
            cv::cuda::GpuMat(input_layer_size.height, input_layer_size.width, CV_8U,
                             &(dst.ptr()[0])),
            cv::cuda::GpuMat(input_layer_size.height, input_layer_size.width, CV_8U,
                             &(dst.ptr()[width])),
            cv::cuda::GpuMat(input_layer_size.height, input_layer_size.width, CV_8U,
                             &(dst.ptr()[width * 2]))
            };

    cv::cuda::split(img, input_channels);
}

bool YOLO::detection(void* input_img_data, const float threshold, const uint32_t max_object)
{
    void* input;
    void* output;
    void* binding[2];
    bool  predict_succese;

    int output_size;
    int num_class;
    int channel;

    int num_block;
    int data_size;

    float*    scores_device;
    int*      class_idxes_device;
    cv::Rect* box_device;

    float*    scores_host;
    int*      class_idxes_host;
    cv::Rect* box_host;

    std::vector<float>    score_vector;
    std::vector<cv::Rect> boxes_vector;
    std::vector<int>      labels_vector;
    std::vector<int>      indices;

    int    output_vector_size;
    int    idx;
    Object obj;

    /*------------------------------------- 입력 데이터 메모리 할당 ------------------------------------------------------*/

    switch(img_datatype)
    {
        case nvinfer1::DataType::kFLOAT:
            data_size = SIZE_FLOAT;
            input = input_img_data;
            break;

        case nvinfer1::DataType::kHALF:
            data_size = SIZE_HALF * input_layer_size.area() * 3;
            hipMalloc(&input, data_size);
            num_block = std::ceil(data_size / MAX_THREAD);
            float2half<<<num_block, MAX_THREAD>>>(static_cast<float*>(input),
                                                  static_cast<half*>(input_img_data),
                                                  data_size);
            break;

        default:
            fprintf(stderr, "YOLO only supports half or float.");
            return false;
    }

    /*------------------------------------- 출력 데이터 메모리 할당 ------------------------------------------------------*/

    // 출력 데이터 크기
    output_size = output0_dims.d[1] * output0_dims.d[2];  // 출력 데이터 크기
    num_class   = output0_dims.d[1] - BOX_IDX;           // 데이터 내 클래스 개수 저장
    channel     = output0_dims.d[2];                         // 데이터 채널 저장

    // 출력 데이터 메모리 할당
    hipMalloc(&output, data_size * output_size);

    /*--------------------------------------------- 추론 --------------------------------------------------------------*/

    // 입출력 바인딩
    binding[input_bind_idx]   = input;
    binding[output0_bind_idx] = output;

    predict_succese = model.run(binding);

    if(img_datatype == nvinfer1::DataType::kHALF) hipFree(input);

    if(!predict_succese)
    {
        fprintf(stderr, "ERROR: Prediction Fail.\n");
        hipFree(output);

        return false;
    }

    /*------------------------------------------- 후처리 --------------------------------------------------------------*/

    // 결과값 변수 선언
    hipMalloc(&scores_device, channel * SIZE_FLOAT);
    hipMalloc(&class_idxes_device, channel * sizeof(int));
    hipMalloc(&box_device, channel * sizeof(cv::Rect));

    num_block = std::ceil(static_cast<float>(channel) / static_cast<float>(MAX_THREAD));

    post_process_detect<<<num_block, MAX_THREAD>>>(output,
                                                   scores_device,
                                                   class_idxes_device,
                                                   box_device,
                                                   num_class, channel, data_size,
                                                   img_size.width, img_size.height, aspect_ratio);
    hipDeviceSynchronize();

    // 출력 배열 초기화
    hipFree(output);

    // GPU 데이터(VRAM)를 HOST(DRAM)로 이동
    scores_host      = static_cast<float*>(malloc(channel * SIZE_FLOAT));
    class_idxes_host = static_cast<int*>(malloc(channel * sizeof(int)));
    box_host         = static_cast<cv::Rect*>(malloc(channel * sizeof(cv::Rect)));

    hipMemcpy(scores_host, scores_device, channel * SIZE_FLOAT, hipMemcpyDeviceToHost);
    hipMemcpy(class_idxes_host, class_idxes_device, channel * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(box_host, box_device, channel * sizeof(cv::Rect), hipMemcpyDeviceToHost);

    // 복사 후 GPU 메모리 해제
    hipFree(scores_device);
    hipFree(class_idxes_device);
    hipFree(box_device);

    // 벡터 데이터로 변경
    score_vector  = std::vector<float>(scores_host, scores_host + channel);
    boxes_vector  = std::vector<cv::Rect>(box_host, box_host + channel);
    labels_vector = std::vector<int>(class_idxes_host, class_idxes_host + channel);

    // 데이터 메모리 해제
    free(scores_host);
    free(class_idxes_host);
    free(box_host);

    // 중첩 영역 감지후 제거
    cv::dnn::NMSBoxesBatched(boxes_vector,
                             score_vector,
                             labels_vector,
                             threshold,
                             NMS_THRESHOLD,
                             indices
                            );

    // 출력 벡터 크기 변경
    output_vector_size = indices.size();

    if(max_object < output_vector_size)
        objects.resize(max_object);
    else
        objects.resize(output_vector_size);

    // 벡터에 데이터 저장
    for(int i = 0; i < objects.size(); i++)
    {
        idx = indices[i];

        obj.bbox  = boxes_vector[idx];
        obj.score = score_vector[idx];
        obj.label = labels_vector[idx];

        objects[i] = obj;
    }

    return true;
}

bool YOLO::segmentation(void* input_img_data, float threshold, uint32_t max_object)
{
    void* input;
    void* output0;
    void* output1;
    void* binding[3];
    bool  predict_succese;

    int output0_size;
    int output1_size;

    int channel;
    int segment_channel;
    int segment_width;
    int segment_height;

    int num_block;
    int data_size;
    int num_class;

    float*    scores_device;
    int*      class_idxes_device;
    cv::Rect* box_device;
    float*    mask_configures;

    float*    scores_host;
    int*      class_idxes_host;
    cv::Rect* box_host;

    std::vector<float>    score_vector;
    std::vector<cv::Rect> boxes_vector;
    std::vector<int>      labels_vector;
    std::vector<int>      indices;

    int output_vector_size;

    int              idx;
    Object           obj{};
    cv::cuda::GpuMat mask_configure;
    cv::cuda::GpuMat mask_device;
    cv::Mat          mask_host;
    cv::Rect         roi;
    cv::cuda::GpuMat prototype;
    float*           output_float;

    /*------------------------------------- 입력 데이터 메모리 할당 ------------------------------------------------------*/

    switch(img_datatype)
    {
        case nvinfer1::DataType::kFLOAT:
            data_size = SIZE_FLOAT;
            input = input_img_data;
            break;

        case nvinfer1::DataType::kHALF:
            data_size = SIZE_HALF;
            hipMalloc(&input, data_size * input_layer_size.area() * 3);
            num_block = std::ceil(static_cast<float>(data_size * input_layer_size.area() * 3) /
                                  static_cast<float>(MAX_THREAD));
            float2half<<<num_block, MAX_THREAD>>>(static_cast<float*>(input_img_data),
                                                  static_cast<half*>(input),
                                                  input_layer_size.area() * 3);
            break;

        default:
            std::cerr << "YOLO only supports half or float." << std::endl;
            return false;
    }

    /*------------------------------------- 출력 데이터 메모리 할당 ------------------------------------------------------*/

    // 출력 데이터 크기
    output0_size = output0_dims.d[1] * output0_dims.d[2];
    output1_size = output1_dims.d[1] * output1_dims.d[2] * output1_dims.d[3];

    channel         = output0_dims.d[2];    // 데이터 채널 저장
    segment_channel = output1_dims.d[1];    // segment 채널 저장
    segment_width   = output1_dims.d[3];    // segment width
    segment_height  = output1_dims.d[2];    // segment height

    num_class = output0_dims.d[1] - segment_channel - BOX_IDX;    // 데이터 내 클래스 개수 저장

    // 출력 데이터 메모리 할당
    hipMalloc(&output0, data_size * output0_size);
    hipMalloc(&output1, data_size * output1_size);

    /*--------------------------------------------- 추론 --------------------------------------------------------------*/

    // 입출력 바인딩
    binding[input_bind_idx]   = input;
    binding[output0_bind_idx] = output0;
    binding[output1_bind_idx] = output1;

    predict_succese = model.run(binding);

    if(img_datatype == nvinfer1::DataType::kHALF) hipFree(input);

    if(!predict_succese)
    {
        fprintf(stderr, "ERROR: Prediction Fail.\n");
        hipFree(output0);
        hipFree(output1);

        return false;
    }

    /*------------------------------------------- 후처리 --------------------------------------------------------------*/

    // 결과값 변수 선언
    hipMalloc(&scores_device, channel * SIZE_FLOAT);
    hipMalloc(&class_idxes_device, channel * sizeof(int));
    hipMalloc(&box_device, channel * sizeof(cv::Rect));
    hipMalloc(&mask_configures, channel * segment_channel * SIZE_FLOAT);

    // 연산에 사용될 블럭 개수 계산
    num_block = std::ceil(static_cast<float>(channel) / static_cast<float>(MAX_THREAD));

    post_process_segment<<<num_block, MAX_THREAD>>>(output0,
                                                    scores_device,
                                                    class_idxes_device,
                                                    box_device,
                                                    mask_configures,
                                                    num_class, channel, segment_channel, data_size,
                                                    img_size.width, img_size.height, aspect_ratio);
    hipDeviceSynchronize();

    hipFree(output0);

    // GPU 데이터(VRAM)를 HOST(DRAM)로 이동
    scores_host      = static_cast<float*>(malloc(channel * SIZE_FLOAT));
    class_idxes_host = static_cast<int*>(malloc(channel * sizeof(int)));
    box_host         = static_cast<cv::Rect*>(malloc(channel * sizeof(cv::Rect)));

    hipMemcpy(scores_host, scores_device, channel * SIZE_FLOAT, hipMemcpyDeviceToHost);
    hipMemcpy(class_idxes_host, class_idxes_device, channel * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(box_host, box_device, channel * sizeof(cv::Rect), hipMemcpyDeviceToHost);

    // 복사 후 GPU 메모리 해제
    hipFree(scores_device);
    hipFree(class_idxes_device);
    hipFree(box_device);

    // 벡터 데이터로 변경
    score_vector  = std::vector<float>(scores_host, scores_host + channel);
    boxes_vector  = std::vector<cv::Rect>(box_host, box_host + channel);
    labels_vector = std::vector<int>(class_idxes_host, class_idxes_host + channel);

    // 데이터 메모리 해제
    free(scores_host);
    free(class_idxes_host);
    free(box_host);

    // 중첩 영역 감지후 제거
    cv::dnn::NMSBoxesBatched(boxes_vector,
                             score_vector,
                             labels_vector,
                             threshold,
                             NMS_THRESHOLD,
                             indices
                            );

    // 출력 벡터 크기 변경
    output_vector_size = indices.size();

    if(max_object < output_vector_size)
        objects.resize(max_object);
    else
        objects.resize(output_vector_size);

    // mask ROI 설정
    if(img_size.height > img_size.width)
        roi = cv::Rect(0, 0, segment_width * img_size.width / img_size.height, segment_height);
    else
        roi = cv::Rect(0, 0, segment_width, segment_height * img_size.height / img_size.width);

    // 프로토타입 저장
    switch(img_datatype)
    {
        case nvinfer1::DataType::kFLOAT:
            prototype = cv::cuda::GpuMat(segment_channel, segment_width * segment_height, CV_32F, output1);
            break;

        case nvinfer1::DataType::kHALF:
            hipMalloc(&output_float, SIZE_FLOAT * output1_size);
            num_block = std::ceil(static_cast<float>(output1_size) / static_cast<float>(MAX_THREAD));
            half2float<<<num_block, MAX_THREAD>>>(static_cast<half*>(output1), output_float, output1_size);
            prototype = cv::cuda::GpuMat(segment_channel, segment_width * segment_height, CV_32F, output_float);

        default:
            break;
    }

    // 벡터에 출력 저장
    for(int i = 0; i < objects.size(); i++)
    {
        idx = indices[i];

        obj.bbox  = boxes_vector[idx];
        obj.score = score_vector[idx];
        obj.label = labels_vector[idx];

        // 세그멘트 마스크 계산
        // 세그멘트 설정값을 행렬에 저장
        mask_configure = cv::cuda::GpuMat(1, segment_channel, CV_32F,
                                          mask_configures + (segment_channel * idx));

        // 설정값과 프로토타입 행렬곱 및 전치
        cv::cuda::gemm(mask_configure, prototype, 1., cv::cuda::GpuMat(), 0.,
                       mask_device, 0.);
        mask_device = mask_device.reshape(1, segment_height);
        mask_device = mask_device(roi);

        cv::cuda::multiply(-1., mask_device, mask_device);
        cv::cuda::exp(mask_device, mask_device);
        cv::cuda::add(1., mask_device, mask_device);
        cv::cuda::divide(1., mask_device, mask_device);

        cv::cuda::resize(mask_device, mask_device, img_size, cv::INTER_LINEAR);
        cv::cuda::compare(mask_device(obj.bbox), threshold, mask_device, cv::CMP_GT);

        mask_device.download(mask_host);

        obj.seg = mask_host;

        objects[i] = obj;
    }

    hipFree(mask_configures);
    hipFree(output1);
    if(img_datatype == nvinfer1::DataType::kHALF) hipFree(output_float);

    return true;
}

void YOLO::draw_segment(const cv::Mat& img, cv::Mat& dst, const Object& object, const cv::Scalar& color)
{
    dst = img.clone();

    cv::Mat color_mat = cv::Mat(img.size(), img.type(), color);
    color_mat(object.bbox).copyTo(color_mat, object.seg);
    cv::addWeighted(img(object.bbox), 1, color_mat, 0.5, 0, dst(object.bbox));
}

std::vector<YOLO::Object> YOLO::get_objects()
{
    return objects;
}
