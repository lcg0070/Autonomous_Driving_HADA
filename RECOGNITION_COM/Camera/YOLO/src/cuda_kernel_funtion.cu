#include "hip/hip_runtime.h"
//
// Created by jarry_goon on 2024-02-26.
//

#include "cuda_kernel_funtion.cuh"

#define CLAMP(X, MIN, MAX) ((X) < (MIN)?(MIN):(((X) > (MAX))?(MAX):(X)))

__global__ void post_process_detect(const void* src, float*          score, int*             class_idxes, cv::Rect* box,
                                    const int   num_class, const int channel, const int      type_size,
                                    const int   img_width, const int img_height, const float aspect_ratio
)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // 인덱스 범위가 실제 행렬 범위를 벗어나는 경우 함수 실행 X
    if(idx >= channel) return;

    float max_score = 0.f;
    float val;

    float x;
    float y;
    float width;
    float height;

    float x0;
    float x1;
    float y0;
    float y1;

    int class_idx = 0;

    if(type_size == 4)
    {
        const float* src_float = static_cast<const float*>(src);

        x      = src_float[idx];
        y      = src_float[channel + idx];
        width  = src_float[channel * 2 + idx];
        height = src_float[channel * 3 + idx];

        for(int i = 0; i < num_class; i++)
        {
            val = src_float[channel * (i + BOX_IDX) + idx];

            if(max_score < val)
            {
                max_score = val;
                class_idx = i;
            }
        }
    }
    else
    {
        const half* src_half = static_cast<const half*>(src);

        x      = __half2float(src_half[idx]);
        y      = __half2float(src_half[channel + idx]);
        width  = __half2float(src_half[channel * 2 + idx]);
        height = __half2float(src_half[channel * 3 + idx]);

        for(int i = 0; i < num_class; i++)
        {
            val = __half2float(src_half[channel * (i + BOX_IDX) + idx]);

            if(max_score < val)
            {
                max_score = val;
                class_idx = i;
            }
        }
    }

    x0 = CLAMP((x - width * 0.5f) * aspect_ratio, 0.f, img_width);
    x1 = CLAMP((x + width * 0.5f) * aspect_ratio, 0.f, img_width);
    y0 = CLAMP((y - height * 0.5f) * aspect_ratio, 0.f, img_height);
    y1 = CLAMP((y + height * 0.5f) * aspect_ratio, 0.f, img_height);

    box[idx].x      = x0;
    box[idx].y      = y0;
    box[idx].width  = x1 - x0;
    box[idx].height = y1 - y0;

    if(box[idx].width == 0) box[idx].width = 1;
    if(box[idx].height == 0) box[idx].height = 1;

    score[idx]       = max_score;
    class_idxes[idx] = class_idx;
}

__global__ void post_process_segment(const void* src, float*          score, int* class_idxes,
                                     cv::Rect*   box, float*          mask_configures,
                                     const int   num_class, const int channel, const int   segment_channel,
                                     const int   type_size, const int img_width, const int img_height,
                                     const float aspect_ratio
)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // 인덱스 범위가 실제 행렬 범위를 벗어나는 경우 함수 실행 X
    if(idx >= channel) return;

    float max_score = 0.f;
    float val;

    float x;
    float y;
    float width;
    float height;

    float x0;
    float x1;
    float y0;
    float y1;

    int class_idx = 0;

    if(type_size == 4)
    {
        const float* src_float = static_cast<const float*>(src);

        x      = src_float[idx];
        y      = src_float[channel + idx];
        width  = src_float[channel * 2 + idx];
        height = src_float[channel * 3 + idx];

        for(int i = 0; i < num_class; i++)
        {
            val = src_float[channel * (i + BOX_IDX) + idx];

            if(max_score < val)
            {
                max_score = val;
                class_idx = i;
            }
        }

        for(int i                                      = 0; i < segment_channel; i++)
            mask_configures[segment_channel * idx + i] = src_float[channel * (BOX_IDX + num_class + i) + idx];
    }
    else
    {
        const half* src_half = static_cast<const half*>(src);

        x      = __half2float(src_half[idx]);
        y      = __half2float(src_half[channel + idx]);
        width  = __half2float(src_half[channel * 2 + idx]);
        height = __half2float(src_half[channel * 3 + idx]);

        for(int i = 0; i < num_class; i++)
        {
            val = __half2float(src_half[channel * (i + BOX_IDX) + idx]);

            if(max_score < val)
            {
                max_score = val;
                class_idx = i;
            }
        }

        for(int i                                      = 0; i < segment_channel; i++)
            mask_configures[segment_channel * idx + i] =
                    __half2float(src_half[channel * (BOX_IDX + num_class + i) + idx]);
    }

    x0 = CLAMP((x - width * 0.5f) * aspect_ratio, 0.f, img_width);
    x1 = CLAMP((x + width * 0.5f) * aspect_ratio, 0.f, img_width);
    y0 = CLAMP((y - height * 0.5f) * aspect_ratio, 0.f, img_height);
    y1 = CLAMP((y + height * 0.5f) * aspect_ratio, 0.f, img_height);

    box[idx].x      = x0;
    box[idx].y      = y0;
    box[idx].width  = x1 - x0;
    box[idx].height = y1 - y0;

    if(box[idx].width == 0) box[idx].width = 1;
    if(box[idx].height == 0) box[idx].height = 1;

    score[idx]       = max_score;
    class_idxes[idx] = class_idx;
}

__global__ void half2float(const half* src, float* dst, const int size)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= size) return;

    dst[idx] = __half2float(src[idx]);
}

__global__ void float2half(const float* src, half* dst, const int size)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= size) return;

    dst[idx] = __float2half(src[idx]);
}
